#include "hip/hip_runtime.h"
/* Compile and run: nvcc -arch=sm_35 -I/usr/local/include/opencv2/ `pkg-config --cflags opencv` -O3 -o histogram histogram.cu -I/usr/local/include/opencv2/ `pkg-config opencv --cflags --libs` -lopencv_gpu -lopencv_core -lopencv_highgui
 */

#include <iostream>
#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudaarithm.hpp"
#include "parse_input.h"

using namespace cv;

int main(int argc, const char* argv[])
{
	if (argc != 2) {
		printf("Error\n");
		return 0;
	}

	char s_append[] = "histogram";
	char input[100] = {0};
	char output_1[100] = {0};
	char output_2[100] = {0};

	int num_tests = 10;
	clock_t gpu_time;
	clock_t cpu_time;

	parse_input(argv, s_append, input, output_1, output_2);
	Mat input_cpu = imread(input, CV_LOAD_IMAGE_COLOR);

	for (int i = 0; i < num_tests; ++i) {
		std::vector<Mat> bgr_planes;
		split(input_cpu, bgr_planes);

		Mat hist_r_cuda, hist_g_cuda, hist_b_cuda, histImg_cuda;

		// create colors channels
		cuda::GpuMat colorGpu_b(bgr_planes[0]);
		cuda::GpuMat colorGpu_g(bgr_planes[1]);
		cuda::GpuMat colorGpu_r(bgr_planes[2]);
		cuda::GpuMat histGpu_r;
		cuda::GpuMat histGpu_g;
		cuda::GpuMat histGpu_b;

		clock_t tStart2 = clock();
		cv::cuda::calcHist(colorGpu_r, histGpu_r);
		cv::cuda::calcHist(colorGpu_g, histGpu_g);
		cv::cuda::calcHist(colorGpu_b, histGpu_b);
		clock_t tEnd2 = clock();

		histGpu_r.download(hist_r_cuda);
		histGpu_g.download(hist_g_cuda);
		histGpu_b.download(hist_b_cuda);

		/* Establish the number of bins */
		int histSize = 256;

		/* Set the ranges (for B,G,R) */
		float range[] = {0, 256};
		const float* histRange = {range};

		bool uniform = true;
		bool accumulate = false;

		Mat b_hist, g_hist, r_hist;

		/* Compute the histograms */
		clock_t tStart3 = clock();
		calcHist(&bgr_planes[0], 1, 0, Mat(), b_hist, 1, &histSize, &histRange, uniform, accumulate);
		calcHist(&bgr_planes[1], 1, 0, Mat(), g_hist, 1, &histSize, &histRange, uniform, accumulate);
		calcHist(&bgr_planes[2], 1, 0, Mat(), r_hist, 1, &histSize, &histRange, uniform, accumulate);
		clock_t tEnd3 = clock();

		/* Draw the histograms for B, G and R */
		//int hist_w = 512; 
		//int hist_h = 400;
		//int bin_w = cvRound((double) hist_w / histSize);

		//Mat histImage(hist_h, hist_w, CV_8UC3, Scalar(255, 255, 255));
		//Mat histImage_cuda(hist_h, hist_w, CV_8UC3, Scalar(255, 255, 255));

		/* Normalize the result to [0, histImage.rows] */
		//normalize(b_hist, b_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat());
		//normalize(g_hist, g_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat());
		//normalize(r_hist, r_hist, 0, histImage.rows, NORM_MINMAX, -1, Mat());

		//normalize(hist_b_cuda, hist_b_cuda, 0, histImage_cuda.rows, NORM_MINMAX, -1, Mat());
		//normalize(hist_g_cuda, hist_g_cuda, 0, histImage_cuda.rows, NORM_MINMAX, -1, Mat());
		//normalize(hist_r_cuda, hist_r_cuda, 0, histImage_cuda.rows, NORM_MINMAX, -1, Mat());

		/* Draw for each channel */
		//for (int i = 1; i < histSize; ++i) {
		//	line(histImage, Point(bin_w * (i - 1), hist_h - cvRound(b_hist.at<float>(i - 1))),
		//			Point(bin_w * (i), hist_h - cvRound(b_hist.at<float>(i))),
		//			Scalar(255, 0, 0), 2, 8, 0);
		//	line(histImage, Point(bin_w * (i - 1), hist_h - cvRound(g_hist.at<float>(i - 1))),
		//			Point(bin_w * (i), hist_h - cvRound(g_hist.at<float>(i))),
		//			Scalar(0, 255, 0), 2, 8, 0);
		//	line(histImage, Point(bin_w*(i-1), hist_h - cvRound(r_hist.at<float>(i - 1))),
		//			Point(bin_w*(i), hist_h - cvRound(r_hist.at<float>(i))),
		//			Scalar(0, 0, 255), 2, 8, 0);

		//	line(histImage_cuda, Point(bin_w * (i - 1), hist_h - cvRound(hist_b_cuda.at<int>(i - 1))),
		//			Point(bin_w * (i), hist_h - cvRound(hist_b_cuda.at<int>(i))),
		//			Scalar(255, 0, 0), 2, 8, 0);
		//	line(histImage_cuda, Point(bin_w * (i - 1), hist_h - cvRound(hist_g_cuda.at<int>(i - 1))),
		//			Point(bin_w * (i), hist_h - cvRound(hist_g_cuda.at<int>(i))),
		//			Scalar(0, 255, 0), 2, 8, 0);
		//	line(histImage_cuda, Point(bin_w*(i-1), hist_h - cvRound(hist_r_cuda.at<int>(i - 1))),
		//			Point(bin_w*(i), hist_h - cvRound(hist_r_cuda.at<int>(i))),
		//			Scalar(0, 0, 255), 2, 8, 0);
		//}

		/* Display */
		//imwrite(output_1, histImage_cuda);
		//imwrite(output_2, histImage);

		colorGpu_r.release();
		colorGpu_g.release();
		colorGpu_b.release();
		histGpu_r.release();
		histGpu_g.release();
		histGpu_b.release();

		if (i == 0) {
			gpu_time = tEnd2 - tStart2;
			cpu_time = tEnd3 - tStart3;
		} else {
			gpu_time += tEnd2 - tStart2;
			cpu_time += tEnd3 - tStart3;
		}
	}

	input_cpu.release();

	printf("Time taken GPU: %.5f s\n", (double) gpu_time / CLOCKS_PER_SEC / num_tests);
	printf("Time taken CPU: %.5f s\n", (double) cpu_time / CLOCKS_PER_SEC / num_tests);
}

#include "hip/hip_runtime.h"
/* Compile and run: nvcc -arch=sm_35 -I/usr/local/include/opencv2/ `pkg-config --cflags opencv` -O3 -o threshold threshold.cu -I/usr/local/include/opencv2/ `pkg-config opencv --cflags --libs` -lopencv_gpu -lopencv_core -lopencv_highgui
 */

#include <iostream>
#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudaarithm.hpp"
#include "parse_input.h"

using namespace cv;

int main(int argc, const char* argv[])
{
	if (argc != 2) {
		printf("Error\n");
		return 0;
	}

	char s_append[] = "threshold";
	char input[100] = {0};
	char output_1[100] = {0};
	char output_2[100] = {0};

	int num_tests = 10;
	clock_t gpu_time;
	clock_t cpu_time;

	parse_input(argv, s_append, input, output_1, output_2);
	Mat input_cpu = imread(input, CV_LOAD_IMAGE_COLOR);

	for (int i = 0; i < num_tests; ++i) {
		Mat output_cpu;
		cuda::GpuMat output_cuda;

		cuda::GpuMat input_cuda(input_cpu);
		clock_t tStart2 = clock();
		cv::cuda::threshold(input_cuda, output_cuda, 128.0, 255.0, CV_THRESH_BINARY);
		clock_t tEnd2 = clock();
		Mat output_cuda_cpu;
		output_cuda.download(output_cuda_cpu);

		clock_t tStart3 = clock();
		cv::threshold(input_cpu, output_cpu, 128.0, 255.0, CV_THRESH_BINARY);
		clock_t tEnd3 = clock();

		//imwrite(output_1, output_cuda_cpu);
		//imwrite(output_2, output_cpu);
		//imshow("Result", input_cpu);
		//cv::waitKey();
		//imshow("Result", output_cuda_cpu);
		//cv::waitKey();
		//imshow("Result", output_cpu);
		//cv::waitKey();

		input_cuda.release();
		output_cuda.release();
		output_cpu.release();

		if (i == 0) {
			gpu_time = tEnd2 - tStart2;
			cpu_time = tEnd3 - tStart3;
		} else {
			gpu_time += tEnd2 - tStart2;
			cpu_time += tEnd3 - tStart3;
		}
	}

	input_cpu.release();

	printf("Time taken GPU: %.5f s\n", (double) gpu_time / CLOCKS_PER_SEC / num_tests);
	printf("Time taken CPU: %.5f s\n", (double) cpu_time / CLOCKS_PER_SEC / num_tests);
}

#include "hip/hip_runtime.h"
/* Compile and run: nvcc -arch=sm_35 -I/usr/local/include/opencv2/ `pkg-config --cflags opencv` -o gaussian_modified gaussian_modified.cu -I/usr/local/include/opencv2/ `pkg-config opencv --cflags --libs` -lopencv_gpu -lopencv_core -lopencv_highgui
 */

#include <iostream>
#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudaarithm.hpp"
#include "parse_input.h"

using namespace cv;

int main(int argc, const char* argv[])
{
	if (argc != 2) {
		printf("Error\n");
		return 0;
	}

	char s_append[] = "gaussian_modified";
	char input[100] = {0};
	char output_1[100] = {0};
	char output_2[100] = {0};

	int num_tests = 10;
	clock_t gpu_time;
	clock_t cpu_time;

	parse_input(argv, s_append, input, output_1, output_2);

	Mat input_cpu = imread(input, CV_LOAD_IMAGE_COLOR);
	Mat output_cpu;
	std::vector<Mat> bgr_planes;
	split(input_cpu, bgr_planes);

	for (int i = 0; i < num_tests; ++i) {
		cuda::GpuMat output_cuda;

		cuda::GpuMat input_cuda(bgr_planes[1]);
		clock_t tStart2 = clock();
		cv::Ptr<cv::cuda::Filter> filter = cv::cuda::createGaussianFilter(input_cuda.type(), output_cuda.type(), Size(3, 3), 0, 0);
		filter->apply(input_cuda, output_cuda);
		clock_t tEnd2 = clock();
		Mat output_cuda_cpu;
		output_cuda.download(output_cuda_cpu);

		clock_t tStart3 = clock();
		cv::GaussianBlur(bgr_planes[1], output_cpu, Size(3, 3), 0, 0);
		clock_t tEnd3 = clock();

		//imwrite(output_1, output_cuda_cpu);
		//imwrite(output_2, output_cpu);
		//imshow("Result", input_cpu);
		//cv::waitKey();
		//imshow("Result", output_cuda_cpu);
		//cv::waitKey();
		//imshow("Result", output_cpu);
		//cv::waitKey();

		input_cuda.release();
		output_cuda.release();
		output_cpu.release();

		if (i == 0) {
			gpu_time = tEnd2 - tStart2;
			cpu_time = tEnd3 - tStart3;
		} else {
			gpu_time += tEnd2 - tStart2;
			cpu_time += tEnd3 - tStart3;
		}
	}

	input_cpu.release();

	printf("Time taken GPU: %.5f s\n", (double) gpu_time / CLOCKS_PER_SEC / num_tests);
	printf("Time taken CPU: %.5f s\n", (double) cpu_time / CLOCKS_PER_SEC / num_tests);
}

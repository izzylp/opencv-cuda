#include "hip/hip_runtime.h"
/* Compile and run: nvcc -arch=sm_35 -I/usr/local/include/opencv2/ `pkg-config --cflags opencv` -O3 -o bilateralFilter bilateralFilter.cu -I/usr/local/include/opencv2/ `pkg-config opencv --cflags --libs` -lopencv_gpu -lopencv_core -lopencv_highgui
 */

#include <iostream>
#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudaarithm.hpp"
#include "parse_input.h"

using namespace cv;

int main(int argc, const char* argv[])
{
	if (argc != 2) {
		printf("Error\n");
		return 0;
	}

	char s_append[] = "bilateralFilter";
	char input[100] = {0};
	char output_1[100] = {0};
	char output_2[100] = {0};

	int num_tests = 10;
	clock_t gpu_time;
	clock_t cpu_time;

	parse_input(argv, s_append, input, output_1, output_2);
	Mat input_cpu = imread(input, CV_LOAD_IMAGE_COLOR);

	for (int i = 0; i < num_tests; ++i) {
		Mat output_cpu;

		cuda::GpuMat output_cuda;

		cuda::GpuMat input_cuda(input_cpu);
		clock_t tStart2 = clock();
		cuda::bilateralFilter(input_cuda, output_cuda, 21, 150, 150);
		clock_t tEnd2 = clock();
		Mat output_cuda_cpu;
		output_cuda.download(output_cuda_cpu);

		clock_t tStart3 = clock();
		bilateralFilter(input_cpu, output_cpu, 21, 150, 150);
		clock_t tEnd3 = clock();

		//imwrite(output_1, output_cuda_cpu);
		//imwrite(output_2, output_cpu);

		input_cuda.release();
		output_cuda.release();
		output_cpu.release();

		if (i == 0) {
			gpu_time = tEnd2 - tStart2;
			cpu_time = tEnd3 - tStart3;
		} else {
			gpu_time += tEnd2 - tStart2;
			cpu_time += tEnd3 - tStart3;
		}
	}
	input_cpu.release();

	printf("Time taken GPU: %.5f s\n", (double) gpu_time / CLOCKS_PER_SEC / num_tests);
	printf("Time taken CPU: %.5f s\n", (double) cpu_time / CLOCKS_PER_SEC / num_tests);
}
